#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include "../timer.cuh"
#include <math.h>
#include <iostream>
#include <fstream>
#include <hiprand.h> //for host
#include <hiprand/hiprand_kernel.h> // for device
#include "../MT.h"
#include <sys/stat.h>
#include <mpi.h>
using namespace std;

//Using "const", the variable is shared into both gpu and cpu. 
const int  NT = 1024; //Num of the cuda threads.
const int  NP = 1e+4; //Particle number.
const int  NB = (NP+NT-1)/NT; //Num of the cuda blocks.
const int  NN = 100;
const int  NPC = 1000; // Number of the particles in the neighbour cell 
const double dt = 0.01;
const int timemax = 5e5;
const int timeeq = 1000;
//Langevin parameters
const double zeta = 1.0;
const double temp = 0.5;
const double rho = 0.80;
const double RCHK= 2.0;
const double rcut= 1.0;


//Initiallization of "hiprandState"
__global__ void setCurand(unsigned long long seed, hiprandState *state){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  hiprand_init(seed, i_global, 0, &state[i_global]);
}

//Gaussian random number's generation
__global__ void genrand_kernel(float *result, hiprandState *state){  
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  result[i_global] = hiprand_normal(&state[i_global]);
}

//Gaussian random number's generation
__global__ void langevin_kernel(double*x_dev,double*y_dev,double *vx_dev,double *vy_dev,double *fx_dev,double *fy_dev,hiprandState *state, double noise_intensity,double LB){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;

  if(i_global<NP){
    vx_dev[i_global] += -zeta*vx_dev[i_global]*dt+ fx_dev[i_global]*dt + noise_intensity*hiprand_normal(&state[i_global]);
    vy_dev[i_global] += -zeta*vy_dev[i_global]*dt+ fy_dev[i_global]*dt + noise_intensity*hiprand_normal(&state[i_global]);
    x_dev[i_global] += vx_dev[i_global]*dt;
    y_dev[i_global] += vy_dev[i_global]*dt;

    x_dev[i_global]  -= LB*floor(x_dev[i_global]/LB);
    y_dev[i_global]  -= LB*floor(y_dev[i_global]/LB);
  }
}



__global__ void disp_gate_kernel(double LB,double *vx_dev,double *vy_dev,double *dx_dev,double *dy_dev,int *gate_dev)
{
  double r2;  
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  
  if(i_global<NP){
    dx_dev[i_global]+=vx_dev[i_global]*dt;
    dy_dev[i_global]+=vy_dev[i_global]*dt;
    r2 = dx_dev[i_global]*dx_dev[i_global]+dy_dev[i_global]*dy_dev[i_global]; //displacement calculation
    if(r2> 0.25*(RCHK-rcut)*(RCHK-rcut)){ //after update list, threshold check!
      gate_dev[0]=1;
    }
  }
}


__global__ void update(double LB,double *x_dev,double *y_dev,double *dx_dev,double *dy_dev,int *list_dev,int *gate_dev)
{
  double dx,dy,r2;  
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  
  if(gate_dev[0] == 1 && i_global<NP){
    
    list_dev[NN*i_global]=0;      
    for (int j=0; j<NP; j++)
      if(j != i_global){
	dx =x_dev[i_global] - x_dev[j];
	dy =y_dev[i_global] - y_dev[j];

	dx -=LB*floor(dx/LB+0.5);
	dy -=LB*floor(dy/LB+0.5);	 

	r2 = dx*dx + dy*dy;

	if(r2 < RCHK*RCHK){
	  list_dev[NN*i_global]++;
	  list_dev[NN*i_global+list_dev[NN*i_global]]=j;
	}
      }
    //    printf("i=%d, list=%d\n",i_global,list_dev[NN*i_global]);      
    dx_dev[i_global]=0.;
    dy_dev[i_global]=0.;
    if(i_global ==0)
      gate_dev[0]=0;
  }
}

__device__ int f(int i,int M){
  int k;
  k=i;
  if(k>=M)
    k-=M;
  if(k<0)
    k+=M;
  return k;
}



__global__ void cell_map(double LB,double *x_dev,double *y_dev,int *map_dev,int *gate_dev, int M)
{
  
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  int nx,ny;
  int num;
  
  if(gate_dev[0] == 1 && i_global<NP){
    
    nx=f((int)(x_dev[i_global]*(double)M/(double)LB),M);
    ny=f((int)(y_dev[i_global]*(double)M/(double)LB),M);
    
    //  for(int m=ny-1;m<=ny+1;m++)
    //  for(int l=nx-1;l<=nx+1;l++){
    num = atomicAdd(&map_dev[(nx+M*ny)*NPC],1); // in map[nx+ny*M][0], count ++, but, If there are more two particle in a box, we must use atomicAdd.
    //atomicAdd : only add integer.
    // num = map_dev[(nx+M*ny)*NPC]+1; first particle at that time, num isn't "1", but "0". so we do "+1" at map_dev.
    // if(num == 0)
    //  printf("%d = %d\n",num,map_dev[(nx+M*ny)*NPC]);
    map_dev[(nx+M*ny)*NPC+num+1] = i_global; // map[nx+ny*M][map[nx+ny*M][0] +1] = k 
    //	if(num>70)
    //	printf("i=%d, map_dev=%d, f=%d, MM=%d, num=%d\n",i_global,map_dev[(f(l,M)+M*f(m,M))*NPC + num], f(l,M)+M*f(m,M),M*M,num);
    // }
    //  printf("i=%d\n",i_global);    
    // }
    //  printf("i=%d, map_dev=%d, f=%d, MM=%d, num=%d\n",i_global,map_dev[(f(l,M)+M*f(m,M))*NPC + num], f(l,M)+M*f(m,M),M*M,num);
  }
}




int calc_com(double *x_corr, double *y_corr, double *corr_x, double *corr_y){
  *corr_x = 0.;
  *corr_y = 0.; 
  for (int i=0; i<NP; i++){
    *corr_x += x_corr[i];
    *corr_y += y_corr[i];
  }
  //printf("%f  %f\n",*corr_x, *corr_y);

  return 0;
}

double calc_MSD(double *MSD_host){
  double msd = 0.;

  for (int i=0; i<NP; i++){
    msd += MSD_host[i];
  }
  return msd;
}

double calc_ISF(double *ISF_host){
  double isf = 0.;

  for (int i=0; i<NP; i++){
    isf += ISF_host[i];
  }

  return isf;
}

double calc_K(double *vx, double*vy){
  double K = 0.;
  for (int i=0; i<NP; i++){
    K += (vx[i]*vx[i]+vy[i]*vy[i])*0.5/(double) NP;
  }
  return K;
}

void save_position(double *x, double*y, double *xi, double *yi, double corr_x, double corr_y){
  for (int i=0; i<NP; i++){
    xi[i] = x[i] - corr_x;
    //if(i%1000==0){printf("%.4f %.4f %.4f\n",*corr_x, x0[i], x[i]);}
    yi[i] = y[i] - corr_y;
  }
  //return 0;
}

__global__ void com_correction(double *x_dev, double *y_dev, double *x_corr_dev, double *y_corr_dev, double LB){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  static double x0[NP], y0[NP];
  static bool IsFirst=true;

  if(i_global<NP){
    if(IsFirst){
      x0[i_global] = x_dev[i_global];
      y0[i_global] = y_dev[i_global];
      IsFirst = false;
    }

    double dx, dy;
    dx = x_dev[i_global] - x0[i_global];
    dy = y_dev[i_global] - y0[i_global];

    dx -= LB*floor(dx/LB+0.5);
    dy -= LB*floor(dy/LB+0.5);

    x_corr_dev[i_global] += dx/NP;
    y_corr_dev[i_global] += dy/NP;
    //if(i_global%1000 == 0){printf("%d %.5f	%.5f\n", i_global, x0[i_global], x_dev[i_global]);}
    x0[i_global] = x_dev[i_global];
    y0[i_global] = y_dev[i_global];
  }
}  
  
__global__ void calculate_rdf(double *x, double *y, double LB, double delta_r,
                   double *r, int ri, double *histogram) {
    int i_global = threadIdx.x + blockIdx.x*blockDim.x; 
    int j;
    if(i_global<NP){
        for (j = 0 ; j < NP; j++) {
            double dx = x[i_global] - x[j];
            double dy = y[i_global] - y[j];
            dx -= LB*floor(dx/LB+0.5);
            dy -= LB*floor(dy/LB+0.5);
            double distance = sqrt(dx * dx + dy * dy);
            int bin_index = (int)(distance / delta_r);
            if (bin_index < ri) {
                histogram[i_global*ri + bin_index] += 1.;
            }
        }
    }
}

__global__ void calculate_structure_factor(double *x_dev, double *y_dev, double LB, double *q_dev, double *Sq_dev, int si){
	int i_global = threadIdx.x + blockIdx.x*blockDim.x; 
	int j;
	double dq = 2.0 * M_PI / LB;
    double cos_sum = 0., sin_sum=0.;
    if(i_global<si){
        q_dev[i_global] = (double) i_global * dq;
        for (j = 0; j<NP; j++){
            double arg = q_dev[i_global] * x_dev[j], arg2 = q_dev[i_global] * y_dev[j];
            cos_sum += cos(arg) + cos(arg2);
            sin_sum += sin(arg) + sin(arg2);
        }
        Sq_dev[i_global] += (cos_sum*cos_sum+sin_sum*sin_sum)/(double)(NP)/2.;
    }
}

__global__ void reduce_rdf(int ri, double *r, double *rdf_dev, double *histogram, double delta_r, int rdf_count)
{    // Calculate RDF
    int i_global = threadIdx.x + blockIdx.x*blockDim.x;
    int k;
    if(i_global<ri){
        r[i_global] = delta_r * (i_global + 0.5);  // Midpoint of the bin
        for (k=0;k<NP;k++){
        rdf_dev[i_global] += histogram[i_global+k*ri]/(2*M_PI*r[i_global]*delta_r*rho*NP)/(double) rdf_count;
        }
	}    
}



__global__ void cell_list(double LB,double *x_dev,double *y_dev,double *dx_dev,double *dy_dev,int *list_dev,int *map_dev,int *gate_dev, int M)
{
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  int nx,ny;
  int j,k;
  double dx,dy,r2;  
  int l,m;
  //  printf("i=%d \n",i_global); 
  if(gate_dev[0] == 1 && i_global<NP){
    // if(i_global==0)
    // printf("update\n");
    list_dev[NN*i_global]=0;
    
    nx=f((int)(x_dev[i_global]*(double)M/(double)LB),M); // what is coordinate of box where the particle is?
    ny=f((int)(y_dev[i_global]*(double)M/(double)LB),M);
    
    for(m=ny-1;m<=ny+1;m++) // x coordinate of box
      for(l=nx-1;l<=nx+1;l++){ // y coordinate of box
        for(k=1; k<=map_dev[(f(l,M)+M*f(m,M))*NPC]; k++){ //NPC = neighbor particle number
          j = map_dev[(f(l,M)+M*f(m,M))*NPC+k]; // don't forget 1-dimensional list
          if(j != i_global){
            dx = x_dev[i_global] - x_dev[j];
            dy = y_dev[i_global] - y_dev[j];
            dx -=LB*floor(dx/LB+0.5);
            dy -=LB*floor(dy/LB+0.5);	  
            r2 = dx*dx + dy*dy;
            if(r2 < RCHK*RCHK){
              list_dev[NN*i_global]++;
              list_dev[NN*i_global+list_dev[NN*i_global]]=j;
              // printf("i=%d, list=%d\n",i_global,list_dev[NN*i_global]);     
            }
          }
        }
      }
    //    printf("i=%d, list=%d\n",i_global,list_dev[NN*i_global]);      
    dx_dev[i_global]=0.;
    dy_dev[i_global]=0.;
    if(i_global==0)
      gate_dev[0]=0;
  } 
}


__global__ void calc_force_BHHP_kernel(double*x_dev,double*y_dev,double *fx_dev,double *fy_dev,double *a_dev,double LB,int *list_dev){
  double dx,dy,dU,a_ij,r2, w2,w4,w12,cut;
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  //a_i  = a_dev[i_global];
  cut = 3.0;
  if(i_global<NP){
    fx_dev[i_global] = 0.0;
    fy_dev[i_global] = 0.0;
    for(int j = 1; j<=list_dev[NN*i_global]; j++){ //list[i][0]
      dx= x_dev[list_dev[NN*i_global+j]] - x_dev[i_global]; //x[list[i][j]-x[i]
      dy= y_dev[list_dev[NN*i_global+j]] - y_dev[i_global];
      
      dx -= LB*floor(dx/LB+0.5);
      dy -= LB*floor(dy/LB+0.5);	
      //dr = sqrt(dx*dx+dy*dy);
      a_ij=0.5*(a_dev[i_global]+a_dev[list_dev[NN*i_global+j]]);  //0.5*(a[i]+a[i][j])
      r2 = dx * dx + dy * dy;
      w2 = a_ij * a_ij / r2;
      w4 = w2*w2;
      w12 = w4*w4*w4;
      if(r2 < cut*cut){ //cut off
	      dU = (-12.0)*w12/r2; //derivertive of U wrt r for harmonic potential.
         fx_dev[i_global] += dU*dx; //only deal for i_global, don't care the for "j"
         fy_dev[i_global] += dU*dy;
      }     
    }
    // printf("i=%d, fx=%f\n",i_global,fx_dev[i_global]);
  }
}


__global__ void calc_force_kernel(double*x_dev,double*y_dev,double *fx_dev,double *fy_dev,double *a_dev,double LB,int *list_dev){
  double dx,dy,dr,dU,a_ij;
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
   
  if(i_global<NP){
    fx_dev[i_global] = 0.;
    fy_dev[i_global] = 0.;
    for(int j = 1; j<=list_dev[NN*i_global]; j++){
      dx=x_dev[list_dev[NN*i_global+j]]-x_dev[i_global];
      dy=y_dev[list_dev[NN*i_global+j]]-y_dev[i_global];
      
      dx -= LB*floor(dx/LB+0.5);
      dy -= LB*floor(dy/LB+0.5);	
      dr = sqrt(dx*dx+dy*dy);
      a_ij = 0.5*(a_dev[i_global]+a_dev[list_dev[NN*i_global+j]]);
      if(dr < a_ij){
	dU = -(1-dr/a_ij)/a_ij; //derivertive of U wrt r.
	fx_dev[i_global] += dU*dx/dr;
	fy_dev[i_global] += dU*dy/dr;
      }      
    }

    // printf("i=%d, fx=%f\n",i_global,fx_dev[i_global]);
  }
}

__global__ void copy_kernel(double *x0_dev, double *y0_dev, double *x_dev, double *y_dev){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  x0_dev[i_global]=x_dev[i_global];
  y0_dev[i_global]=y_dev[i_global];
  // printf("%f,%f\n",x_dev[i_global],x0_dev[i_global]);
}

__global__ void copy_kernel2(double *xi_dev, double *yi_dev, double *x_dev, double *y_dev, double *corr_x_dev, double *corr_y_dev){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  if(i_global<NP){
    xi_dev[i_global] = x_dev[i_global] - *corr_x_dev;
    yi_dev[i_global] = y_dev[i_global] - *corr_y_dev;
    //if(i_global%1000==0){printf("%d, %f, %f\n",i_global, *corr_x_dev,*corr_y_dev);}
  } 
}

__global__ void init_gate_kernel(int *gate_dev, int c){
  gate_dev[0]=c;
}

__global__ void init_map_kernel(int *map_dev,int M){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  // for(int i=0;i<M;i++)
  //  for(int j=0;j<M;j++)
  // map_dev[(i+M*j)*NPC] = 0;
  map_dev[i_global] = 0;
}

__global__ void init_array(double *x_dev, double c){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  x_dev[i_global] = c;
}

__global__ void init_binary(double *x_dev, double c, double c2){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  if(i_global < NP){ 
    if(i_global%2==0){x_dev[i_global] =c;}
    else {x_dev[i_global] =c2;}
  }
}

__global__ void init_array_rand(double *x_dev, double c,hiprandState *state){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  x_dev[i_global] = c*hiprand_uniform(&state[i_global]);
}

void output(double *x,double *y,double *vx,double *vy,double *a){
  static int count=1;
  char filename[128];
  sprintf(filename,"coord_%.d.dat",count);
  ofstream file;
  file.open(filename);
  double temp0=0.0;
  
  for(int i=0;i<NP;i++){
    file << x[i] << " " << y[i]<< " " << a[i] << endl;
    temp0+= 0.5*(vx[i]*vx[i]+vy[i]*vy[i]);
    // cout <<i<<" "<<map[i]<<endl;
  }

  file.close();

  cout<<"temp="<< temp0/NP <<endl;
  count++;
}


__global__ void MSD_ISF_device(double *x_dev, double *y_dev, double *xi_dev, double *yi_dev, double *corr_x_dev, double *corr_y_dev, double *MSD_dev, double *ISF_dev, double LB){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  double dx, dy;
  double q = 2. * M_PI / 1.0;

  if (i_global<NP){
     dx = x_dev[i_global] - xi_dev[i_global] - *corr_x_dev;
     dy = y_dev[i_global] - yi_dev[i_global] - *corr_y_dev;
    //if(i_global%1000==0){printf("%d  %.3f %.3f\n", i_global, dx, dy);}
     dx -= LB*floor(dx/LB+0.5); //boudary condition
     dy -= LB*floor(dy/LB+0.5);	  
     
     MSD_dev[i_global] = (dx*dx + dy*dy)/(double)NP;
     ISF_dev[i_global] = (cos(- q * dx) + cos(- q * dy)) / (double)NP / 2.0;
     //if(i_global%1000==0){printf("%d	%.4f\n",i_global, *corr_x_dev);}
  }
}

void output_Measure(double *measure_time, double *MSD, double *ISF, double *count, int time_count, int eq_count, int ri, double *r, double *rdf_host, int si, double *q_host, double *Sq_host, int rdf_count){
  char filename[128], filename2[128], filename3[128];
  mkdir("data",0755);
  sprintf(filename,"data/MSD_ISF_MPI_T=%.4f.dat",temp);
  FILE *fp,*fp2, *fp3;
  fp = fopen(filename, "w+");
  for(int i=1;i<time_count;i++){
    fprintf(fp, "%.4f\t%.4f\t%.4f\n", measure_time[i]-measure_time[0], MSD[i]/(count[i]-eq_count), ISF[i]/(count[i]-eq_count));
  }
  fclose(fp);
    
  sprintf(filename2,"data/rdf_MPI_T=%.4f.dat",temp);
  fp2 = fopen(filename2, "w+");
  for(int i=1;i<ri;i++){
    fprintf(fp2, "%.4f\t%.4f\n", r[i], rdf_host[i]);
  }
  fclose(fp2);

  sprintf(filename3,"data/Sq_MPI_T=%.4f.dat",temp);
  fp3 = fopen(filename3, "w+");
  for(int i=1;i<si;i++){
    fprintf(fp3, "%.4f\t%.4f\n", q_host[i], Sq_host[i]/(double)rdf_count);
  }
  fclose(fp3);
}


int main(int argc, char** argv){
  double *x,*xi,*xi_dev,*vx,*y,*yi,*yi_dev,*vy,*a,*x_dev,*vx_dev,*y_dev,*dx_dev,*dy_dev,*vy_dev,*a_dev,*fx_dev,*fy_dev;
  double *x_corr_dev, *y_corr_dev, *x_corr, *y_corr, corr_x=0., corr_y=0., *corr_x_dev, *corr_y_dev;
  int *list_dev,*map_dev,*gate_dev, time_count, init_count;
  double *MSD_dev, *MSD_host, *ISF_dev,*ISF_host;
  int ri=1000, rdf_count=0, si = 500;
  double delta_r = 0.01;
  double *histogram, *rdf_dev, *rdf_host, *r_dev, *r_host;
  double *Sq_dev, *Sq_host, *q_dev, *q_host;
  double Sq_MPI[si], rdf_MPI[ri];
  double sampling_time, time_stamp=0.;
  double sampling_time_max =2e+4;
  hiprandState *state; //Cuda state for random numbers
  double sec; //measurred time
  double noise_intensity = sqrt(2.*zeta*temp*dt); //Langevin noise intensity.   
  double LB = sqrt((double)NP/rho);//box length by number fraction
  int M = (int)(LB/RCHK);
  int np, myrank;
  //cout <<M<<endl;
  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MPI_COMM_WORLD,&myrank);
  MPI_Comm_size(MPI_COMM_WORLD, &np);

  int gpu_id = myrank;
  hipSetDevice(gpu_id); 

  x  = (double*)malloc(NB*NT*sizeof(double));
  xi  = (double*)malloc(NB*NT*sizeof(double));
  y  = (double*)malloc(NB*NT*sizeof(double));
  yi  = (double*)malloc(NB*NT*sizeof(double));
  vx = (double*)malloc(NB*NT*sizeof(double));
  vy = (double*)malloc(NB*NT*sizeof(double));
  a  = (double*)malloc(NB*NT*sizeof(double));
  // map  = (int*)malloc(M*M*NPC*sizeof(int));
  x_corr  = (double*)malloc(NB*NT*sizeof(double));
  y_corr  = (double*)malloc(NB*NT*sizeof(double));
  MSD_host  = (double*)malloc(NB*NT*sizeof(double));
  ISF_host  = (double*)malloc(NB*NT*sizeof(double));
  rdf_host  = (double*)malloc(NB*NT*sizeof(double));
  r_host  = (double*)malloc(NB*NT*sizeof(double));
  Sq_host  = (double*)malloc(NB*NT*sizeof(double));
  q_host  = (double*)malloc(NB*NT*sizeof(double));
  hipMalloc((void**)&x_dev,  NB * NT * sizeof(double)); // CudaMalloc should be executed once in the host. 
  hipMalloc((void**)&y_dev,  NB * NT * sizeof(double)); 
  hipMalloc((void**)&xi_dev,  NB * NT * sizeof(double)); 
  hipMalloc((void**)&yi_dev,  NB * NT * sizeof(double)); 
  hipMalloc((void**)&dx_dev,  NB * NT * sizeof(double)); 
  hipMalloc((void**)&dy_dev,  NB * NT * sizeof(double)); 
  hipMalloc((void**)&vx_dev, NB * NT * sizeof(double)); 
  hipMalloc((void**)&vy_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&fx_dev, NB * NT * sizeof(double)); 
  hipMalloc((void**)&fy_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&x_corr_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&y_corr_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&corr_x_dev, sizeof(double));
  hipMalloc((void**)&corr_y_dev, sizeof(double));
  hipMalloc((void**)&a_dev,  NB * NT * sizeof(double)); 
  hipMalloc((void**)&MSD_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&ISF_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&rdf_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&histogram, ri * NP * sizeof(double));
  hipMalloc((void**)&r_dev,  NB * NT * sizeof(hiprandState)); 
  hipMalloc((void**)&Sq_dev,  NB * NT * sizeof(hiprandState));
  hipMalloc((void**)&q_dev,  NB * NT * sizeof(hiprandState));
  hipMalloc((void**)&gate_dev, sizeof(int)); 
  hipMalloc((void**)&list_dev,  NB * NT * NN* sizeof(int)); 
  hipMalloc((void**)&map_dev,  M * M * NPC* sizeof(int)); 
  hipMalloc((void**)&state,  NB * NT * sizeof(hiprandState)); 
  
  sampling_time = 5.*dt;
  time_count = 0;

  for(double t=dt;t<timemax;t+=dt){
    if(int(t/dt)== int((sampling_time + time_stamp)/dt)){
	    sampling_time *=pow(10,0.1);
	    sampling_time=int(sampling_time/dt)*dt;
	    time_count++;
	    //printf("%.5f	%d\n",t, time_count);
	  if(sampling_time > sampling_time_max/pow(10.,0.1)){
	    time_stamp=0.;
	    sampling_time=5.*dt;
	    break;
      }
    }
  } 
  

  int max_count = time_count;
  double measure_time[time_count], MSD[time_count], count[time_count], ISF[time_count], MSD_MPI[time_count], ISF_MPI[time_count];
    //Make the measure time table
    time_count = 0.;
    for(double t=dt;t<timemax;t+=dt){
      if(int(t/dt)== int((sampling_time + time_stamp)/dt)){
        count[time_count] = 0.;
        MSD[time_count] = 0.;
        ISF[time_count] = 0.;
        measure_time[time_count] = t - time_stamp;
        sampling_time *=pow(10,0.1);
        sampling_time=int(sampling_time/dt)*dt;
        printf("%.5f	%d\n", measure_time[time_count], time_count);
        time_count++;
    if(sampling_time > sampling_time_max/pow(10.,0.1)){
      time_stamp=0.;//reset the time stamp
      sampling_time=5.*dt; //reset the time sampling_time
      break;
        }
      }
    }
  
  int rn_seed = rand()+myrank; 
  setCurand<<<NB,NT>>>(rn_seed, state); // Construction of the cudarand state.  

  init_array_rand<<<NB,NT>>>(x_dev,LB,state);
  init_array_rand<<<NB,NT>>>(y_dev,LB,state);
  init_binary<<<NB,NT>>>(a_dev,1.0, 1.4);
  init_array<<<NB,NT>>>(vx_dev,0.);
  init_array<<<NB,NT>>>(vy_dev,0.);
  init_array<<<NB,NT>>>(x_corr_dev,0.);
  init_array<<<NB,NT>>>(y_corr_dev,0.);
  init_array<<<NB,NT>>>(ISF_dev,0.);
  init_array<<<NB,NT>>>(MSD_dev,0.);
  init_array<<<NB,NT>>>(Sq_dev,0.);
  init_array<<<NP,ri>>>(histogram,0.);
  init_gate_kernel<<<1,1>>>(gate_dev,1);
  init_map_kernel<<<M*M,NPC>>>(map_dev,M);
  cell_map<<<NB,NT>>>(LB,x_dev,y_dev,map_dev,gate_dev,M);
  // hipMemcpy(map,map_dev, M * M * NPC* sizeof(int),hipMemcpyDeviceToHost);
  cell_list<<<NB,NT>>>(LB,x_dev,y_dev,dx_dev,dy_dev,list_dev,map_dev,gate_dev,M);
  // hipDeviceSynchronize(); 
  //  update<<<NB,NT>>>(LB,x_dev,y_dev,dx_dev,dy_dev,list_dev,gate_dev);

  measureTime(); 

  for(double t=0;t<timeeq;t+=dt){
    // cout<<t<<endl;
    calc_force_kernel<<<NB,NT>>>(x_dev,y_dev,fx_dev,fy_dev,a_dev,LB,list_dev);
    langevin_kernel<<<NB,NT>>>(x_dev,y_dev,vx_dev,vy_dev,fx_dev,fy_dev,state,0.0,LB);
    disp_gate_kernel<<<NB,NT>>>(LB,vx_dev,vy_dev,dx_dev,dy_dev,gate_dev); //for auto-list method
    init_map_kernel<<<M*M,NPC>>>(map_dev,M);
    cell_map<<<NB,NT>>>(LB,x_dev,y_dev,map_dev,gate_dev,M);
    cell_list<<<NB,NT>>>(LB,x_dev,y_dev,dx_dev,dy_dev,list_dev,map_dev,gate_dev,M);
  }
  
  time_count = 0;
  init_count = 0;
  int eq_count = 10;


  for(double t=dt;t<timemax;t+=dt){
    // cout<<t<<endl;
    calc_force_BHHP_kernel<<<NB,NT>>>(x_dev,y_dev,fx_dev,fy_dev,a_dev,LB,list_dev);
    langevin_kernel<<<NB,NT>>>(x_dev,y_dev,vx_dev,vy_dev,fx_dev,fy_dev,state,noise_intensity,LB);
    //init_gate_kernel<<<1,1>>>(gate_dev,0);
    com_correction<<<NB,NT>>>(x_dev, y_dev, x_corr_dev, y_corr_dev, LB);
    if(int(t/dt)== int((sampling_time + time_stamp)/dt)){
	  count[time_count]++;//measure count at each logarithmic times
            //hipDeviceSynchronize();
      if(init_count>=eq_count){
 		hipMemcpy(x_corr, x_corr_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
      	hipMemcpy(y_corr, y_corr_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
      	calc_com(x_corr, y_corr, &corr_x, &corr_y);
      	hipMemcpy(corr_x_dev, &corr_x, sizeof(double),hipMemcpyHostToDevice);
      	hipMemcpy(corr_y_dev, &corr_y, sizeof(double),hipMemcpyHostToDevice);
       if(time_count==0){
          //hipDeviceSynchronize();
          copy_kernel2<<<NB,NT>>>(xi_dev, yi_dev, x_dev,y_dev, corr_x_dev, corr_y_dev);
        }
        MSD_ISF_device<<<NB,NT>>>(x_dev, y_dev, xi_dev, yi_dev, corr_x_dev, corr_y_dev, MSD_dev, ISF_dev, LB);
        //ISF_device<<<NB,NT>>>(x_dev, y_dev, xi_dev, yi_dev, corr_x_dev, corr_y_dev, ISF_dev, LB);
        hipMemcpy(MSD_host, MSD_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
        hipMemcpy(ISF_host, ISF_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);

        double MSD_temp = calc_MSD(MSD_host); //the variable for check in real-time
        double ISF_temp = calc_ISF(ISF_host); //If you don't need to check, using just sub-routines
        
        MSD[time_count] += MSD_temp;//reduce the MSD from each particles
        ISF[time_count] += ISF_temp; //reduce the ISF from each particles

        printf("%d %d	%.4f	%.4f  %.4f  %.4f  %.4f\n", time_count, init_count, measure_time[time_count], MSD_temp, ISF_temp, corr_x, corr_y);
        //hipMemcpy(vx, vx_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
        //hipMemcpy(vy, vy_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
        //double K = calc_K(vx,vy);
        //output_t(x, y, t, time_stamp, corr_x, corr_y, K);

      }

      else {printf("time = %.4f\n", measure_time[time_count]);}
	    sampling_time *=pow(10,0.1);
	    sampling_time=int(sampling_time/dt)*dt;
	    time_count++;
      
	    if(sampling_time > sampling_time_max/pow(10.,0.1)){
	      time_stamp=t; //memory of initial measure time for logarithmic sampling
	      sampling_time=5.*dt; //reset the time sampling_time
	      init_count++;
        time_count = 0;
      }
    }
    if(int(t/dt)%2000==0){
        calculate_rdf<<<NB,NT>>>(x_dev, y_dev, LB, delta_r, r_dev, ri, histogram);
        calculate_structure_factor<<<NB,NT>>>(x_dev, y_dev, LB, q_dev, Sq_dev, si);
        rdf_count++;
    }

    disp_gate_kernel<<<NB,NT>>>(LB,vx_dev,vy_dev,dx_dev,dy_dev,gate_dev); //max displacement for each particle
    init_map_kernel<<<M*M,NPC>>>(map_dev,M);
    // hipDeviceSynchronize(); // for printf in the device.
    cell_map<<<NB,NT>>>(LB,x_dev,y_dev,map_dev,gate_dev,M);
    cell_list<<<NB,NT>>>(LB,x_dev,y_dev,dx_dev,dy_dev,list_dev,map_dev,gate_dev,M);
  } 

  sec = measureTime()/1000.;
  cout<<"time(sec):"<<sec<<endl;
  MPI_Barrier(MPI_COMM_WORLD);
  MPI_Reduce(&MSD,&MSD_MPI,max_count,MPI_DOUBLE,MPI_SUM,0,MPI_COMM_WORLD);
  MPI_Barrier(MPI_COMM_WORLD);
  MPI_Reduce(&ISF,&ISF_MPI,max_count,MPI_DOUBLE,MPI_SUM,0,MPI_COMM_WORLD);
  MPI_Barrier(MPI_COMM_WORLD);

  if(myrank==0){
  for (int i=0; i<max_count;i++){
	MSD[i] = MSD_MPI[i]/(double)np;
	ISF[i] = ISF_MPI[i]/(double)np;
   }
  }
  MPI_Barrier(MPI_COMM_WORLD);
  //hipMemcpy(x,   x_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
  //hipMemcpy(vx, vx_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
  //hipMemcpy(y,   y_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
  //hipMemcpy(vy, vy_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
  //hipMemcpy(a, a_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
  
  //output(x,y,vx,vy,a);
  reduce_rdf<<<NB,NT>>>(ri,r_dev,rdf_dev,histogram, delta_r, rdf_count);
  hipMemcpy(rdf_host, rdf_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(r_host, r_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(Sq_host, Sq_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(q_host, q_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
  

  MPI_Barrier(MPI_COMM_WORLD);
  MPI_Reduce(Sq_host,&Sq_MPI,si,MPI_DOUBLE,MPI_SUM,0,MPI_COMM_WORLD);
  MPI_Barrier(MPI_COMM_WORLD);
  MPI_Reduce(rdf_host,&rdf_MPI,ri,MPI_DOUBLE,MPI_SUM,0,MPI_COMM_WORLD);
  MPI_Barrier(MPI_COMM_WORLD);

  if(myrank==0){
    for (int i=0; i<ri;i++){
        rdf_host[i] = rdf_MPI[i]/(double)np;
    }
    for (int i=0; i<si;i++){
	    Sq_host[i] = Sq_MPI[i]/(double)np;
   }

  output_Measure(measure_time, MSD, ISF, count, max_count, eq_count, ri, r_host, rdf_host, si, q_host, Sq_host, rdf_count);
  }
  
  
  MPI_Barrier(MPI_COMM_WORLD);
  hipFree(x_dev);
  hipFree(xi_dev);
  hipFree(vx_dev);
  hipFree(y_dev);
  hipFree(yi_dev);
  hipFree(vy_dev);
  hipFree(dx_dev);
  hipFree(dy_dev);
  hipFree(x_corr_dev);
  hipFree(y_corr_dev);
  hipFree(corr_x_dev);
  hipFree(corr_y_dev);
  hipFree(MSD_dev);
  hipFree(ISF_dev);
  hipFree(rdf_dev);
  hipFree(histogram);
  hipFree(r_dev);
  hipFree(q_dev);
  hipFree(Sq_dev);
  hipFree(gate_dev);
  hipFree(state);
  free(x); 
  free(xi); 
  free(vx); 
  free(y); 
  free(yi); 
  free(a); 
  free(x_corr); 
  free(y_corr); 
  free(MSD_host); 
  free(ISF_host);
  free(rdf_host);
  free(r_host);
  free(Sq_host);
  free(q_host);
  free(vy); 
  MPI_Finalize(); 
  return 0;
}
